#include "hip/hip_runtime.h"
#include "SurfacePoseEstimationCuda.h"

namespace Cuda
{
    static float* gpu_query;
    static float* gpu_data;
    static int* gpu_idxs;
    static float* gpu_dist_sq;

    void CheckCUDAError(const char* msg) {
        hipError_t err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    __global__ void Search(const float* query, int query_pts, const float* data, int data_pts,
        int* idxs, float* dist_sq)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= query_pts)
            return;

        int best_idx = -1;
        float best_dist = FLT_MAX;

        for (int i = 0; i < data_pts; i++) {
            float dist_sq = 0;

            for (int j = 0; j < CUDA_NN_DIM; j++) {
                float d = query[idx * CUDA_NN_DIM + j] - data[i * CUDA_NN_DIM + j];
                dist_sq += d * d;
            }

            if (dist_sq < best_dist) {
                best_dist = dist_sq;
                best_idx = i;
            }
        }

        idxs[idx] = best_idx;
        dist_sq[idx] = best_dist;
    }

    void CUDA_NN_Search(const float* query, int query_pts, const float* data, int data_pts,
        int* idxs, float* dist_sq)
    {
        int threads = 256;
        int blocks = query_pts / threads + ((query_pts % threads) ? 1 : 0);

        hipMalloc((void**)&gpu_data, sizeof(float) * data_pts * CUDA_NN_DIM);
        hipMalloc((void**)&gpu_query, sizeof(float) * query_pts * CUDA_NN_DIM);
        hipMalloc((void**)&gpu_idxs, sizeof(int) * query_pts);
        hipMalloc((void**)&gpu_dist_sq, sizeof(float) * query_pts);

        CheckCUDAError("initilisation");

        hipMemcpy(gpu_query, query, sizeof(float) * query_pts * CUDA_NN_DIM, hipMemcpyHostToDevice);
        hipMemcpy(gpu_data, data, sizeof(float) * data_pts * CUDA_NN_DIM, hipMemcpyHostToDevice);

        CheckCUDAError("memory copying");

        Search << <blocks, threads >> > (gpu_query, query_pts, gpu_data, data_pts, gpu_idxs, gpu_dist_sq);
        hipDeviceSynchronize();

        hipMemcpy(idxs, gpu_idxs, sizeof(int) * query_pts, hipMemcpyDeviceToHost);
        hipMemcpy(dist_sq, gpu_dist_sq, sizeof(float) * query_pts, hipMemcpyDeviceToHost);

        hipFree(gpu_query);
        hipFree(gpu_data);
        hipFree(gpu_idxs);
        hipFree(gpu_dist_sq);
    }
}